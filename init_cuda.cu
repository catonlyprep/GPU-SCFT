#include "hip/hip_runtime.h"

#include"struct.h"

#include "init_cuda.h"

#include "cuda_aid.cuh"
#include <complex.h>

extern void init_cuda(GPU_INFO *gpu_info,int display){
	
	int gpu_count;
	//hipDeviceProp_t prop[64];
	int *gpuid;
	int i;
	gpu_count=0;
	
	gpuid=(int*)malloc(sizeof(int));

	
	//checkCudaErrors(hipGetDeviceCount(&gpu_info->GPU_N));

	//if(gpu_info->GPU_N==8) gpu_info->GPU_N=4;//! Set the number of GPU

		
	
	gpu_info->GPU_list=(int*)malloc(sizeof(int)*(gpu_info->GPU_N));

	for(i=0;i<(gpu_info->GPU_N);i++) {
		gpu_info->GPU_list[i]=i;}	//!Define on these GPU to calculate 

	int dev_indx[gpu_info->GPU_N];
        // user defined GPU device index, check which GPU to use by type "nvidia-smi"
        assert(gpu_info->GPU_N==1);
        dev_indx[0]=1;

	for (i=0; i < gpu_info->GPU_N; i++){
		
        	checkCudaErrors(hipGetDeviceProperties(&gpu_info->prop[i], dev_indx[i])); // get the device properties for the specified device number i
		
		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[i])); // cuda runtime API, thread-safe, to select which GPU to execute CUDA calls on
		
		// Only boards based on Fermi can support P2P
		
            	gpuid[gpu_count++] = gpu_info->GPU_list[i];
                printf("Device Number: %d\n", i);
                printf("  Device name: %s\n", gpu_info->prop[i].name);
                printf("  Memory Clock Rate (KHz): %d\n",\
                       gpu_info->prop[i].memoryClockRate);
                printf("  Memory Bus Width (bits): %d\n",\
                       gpu_info->prop[i].memoryBusWidth);
                printf("  Peak Memory Bandwidth (GB/s): %f\n\n",\
                       2.0*gpu_info->prop[i].memoryClockRate*(gpu_info->prop[i].memoryBusWidth/8)/1.0e6);

                printf("Total global memory:           %lu\n",  gpu_info->prop[i].totalGlobalMem);
                printf("Total shared memory per block: %lu\n",  gpu_info->prop[i].sharedMemPerBlock);
                printf("Total registers per block:     %d\n",  gpu_info->prop[i].regsPerBlock);
                printf("Warp size:                     %d\n",  gpu_info->prop[i].warpSize);
                printf("Maximum memory pitch:          %lu\n",  gpu_info->prop[i].memPitch);
                printf("Maximum threads per block:     %d\n",  gpu_info->prop[i].maxThreadsPerBlock);
                for (int j = 0; j < 3; ++j)
                    printf("Maximum dimension %d of block:  %d\n", j, gpu_info->prop[i].maxThreadsDim[j]);
                for (int j = 0; j < 3; ++j)
                    printf("Maximum dimension %d of grid:   %d\n", j, gpu_info->prop[i].maxGridSize[j]);



		
		//gpu_info->thread=gpu_info->prop[i].maxThreadsPerBlock;
		
}

}


extern void initialize_cufft(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){

	
	int Dim[3];
	int i;
	int rank = 3;
	int Nx=cufft_info->Nx;
	int Ny=cufft_info->Ny;
	int Nz=cufft_info->Nz;
	
	long NxNyNz=Nx*Ny*Nz;  //,ijk;
	
	cufft_info->NxNyNz=NxNyNz;
	cufft_info->Nxh1=Nx/2+1; // R2C, first dimension is cut in half for reduandency
	cufft_info->Nxh1NyNz=cufft_info->Nxh1*Ny*Nz; // only near half size of grid in complex fft space
	int batch=1;
	
        // doing the factor decompose to determine the thread grid dimension?
	//printf("gpu_info->thread_sur %d\n",gpu_info->thread_sur);
	
	//char comment[200];

	
	//!----------- Initialize GPU memery settings. ------------------------------------------------------	
	
	//int nGPUs = gpu_info->GPU_N;
	
	cufft_info->device_in.resize(gpu_info->GPU_N);
	cufft_info->device_out.resize(gpu_info->GPU_N);
	//-----------! Initialize CUFFT settings. ------------------------------------------------------
	
	dim3 grid(cufft_info->Nx,cufft_info->Ny,cufft_info->Nz),block(1,1,1); // set the grid dimension as (Nx,Ny,Nz). one thread per block
	
	Dim[0]=Nz;Dim[1]=Ny;Dim[2]=Nx;

	cufft_info->plan_forward=(hipfftHandle *)malloc(sizeof(hipfftHandle)*gpu_info->GPU_N);
	cufft_info->plan_backward=(hipfftHandle *)malloc(sizeof(hipfftHandle)*gpu_info->GPU_N);

	for(int gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
		
		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[gpu_index]));
	
		checkCudaErrors(hipfftCreate(&cufft_info->plan_forward[gpu_index]));
		checkCudaErrors(hipfftCreate(&cufft_info->plan_backward[gpu_index]));
		
		
		if(rank==3){
			
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_forward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_D2Z, batch));
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_backward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_Z2D, batch));
		
		}
		else if(rank==2) {
		
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_forward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_D2Z, batch));
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_backward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_Z2D, batch));

		}
	}
	
	
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed [  ]");
	printf("Wonderful We have successfully initialized cufft setting.\n");

	//-----------! Initialize malloc and initilize on CPU. ------------------------------------------------------	
	gpu_info->stream=(hipStream_t*)malloc( sizeof(hipStream_t)*gpu_info->GPU_N);
	
	
	printf("Wonderful We have successfully initialized CPU setting.\n");
	
	//-----------! Initialize malloc and initilize on each GPUs. ------------------------------------------------------	

	for (i=0; i < gpu_info->GPU_N; i++){

		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[i]));
		checkCudaErrors(hipStreamCreate(&gpu_info->stream[i]));
		checkCudaErrors(hipfftSetStream(cufft_info->plan_forward[i], gpu_info->stream[i]));
		checkCudaErrors(hipfftSetStream(cufft_info->plan_backward[i], gpu_info->stream[i]));
		
	//	checkCudaErrors(hipMallocManaged((void**)&(cufft_info->kxyzdz_cu[i]), sizeof(double)* NxNyNz));
		checkCudaErrors(hipMalloc(&(cufft_info->device_in[i]), sizeof(double)* cufft_info->NxNyNz*batch));
		checkCudaErrors(hipMalloc(&(cufft_info->device_out[i]), sizeof(hipfftDoubleComplex)* cufft_info->Nxh1NyNz*batch));

		checkCudaErrors(hipDeviceSynchronize());
		
		
	}
	
	
	
	printf("Wonderful We have successfully initialized all the data.\n");
	
	
}


extern void init_chain_chemical(GPU_INFO *gpu_info,GRID *grid,CELL *cell,CHEMICAL *chemical,CHAIN *chain) {
        double *kx,*ky,*kz;  
        double dx,dy,dz,ksq;
        int Nx,Ny,Nz,i,j,k;
        long NxNyNz,ijk;
        Nx=grid->Nx;   
        Ny=grid->Ny;   
        Nz=grid->Nz;   
        NxNyNz=Nx*Nx*Nz;   
	kx=(double *)malloc(sizeof(double)*Nx);
	ky=(double *)malloc(sizeof(double)*Ny);
	kz=(double *)malloc(sizeof(double)*Nz);
	dx=cell->dx;
	dy=cell->dy;
	dz=cell->dz;
	
	chain->exp_ksq=(double *)malloc(sizeof(double)*NxNyNz);	
	chemical->exp_w=(double *)malloc(sizeof(double)*NxNyNz*chemical->N_spe);	
	for(i=0;i<=Nx/2-1;i++) kx[i]=2*Pi*i*1.0/Nx/dx;
	for(i=Nx/2;i<Nx;i++)   kx[i]=2*Pi*(i-Nx)*1.0/dx/Nx;
	for(i=0;i<Nx;i++)      kx[i]*=kx[i];

	for(i=0;i<=Ny/2-1;i++) ky[i]=2*Pi*i*1.0/Ny/dy;
	for(i=Ny/2;i<Ny;i++)   ky[i]=2*Pi*(i-Ny)*1.0/dy/Ny;
	for(i=0;i<Ny;i++)      ky[i]*=ky[i];

	for(i=0;i<=Nz/2-1;i++) kz[i]=2*Pi*i*1.0/Nz/dz;
	for(i=Nz/2;i<Nz;i++)   kz[i]=2*Pi*(i-Nz)*1.0/dz/Nz;
	for(i=0;i<Nz;i++)      kz[i]*=kz[i];
	double ds;
        ds=1.0/chain->Ns;
	for(k=0;k<Nz;k++) {
	   for(j=0;j<Ny;j++){
	      for(i=0;i<Nx;i++){
		ijk=(long)((k*Ny+j)*Nx+i);// x is the fastest dimension!!
		ksq=kx[i]+ky[j]+kz[k];
		chain->exp_ksq[ijk]=exp(-ds*ksq);
	                       }
                             }
                           }
	
	checkCudaErrors(hipMallocManaged(&chain->exp_ksq_cu, sizeof(double)* NxNyNz));
	checkCudaErrors(hipMallocManaged(&chemical->exp_w_cu, sizeof(double)* NxNyNz*chemical->N_spe));
	
	checkCudaErrors(hipMemcpy(chain->exp_ksq_cu, chain->exp_ksq,sizeof(double)*NxNyNz,hipMemcpyHostToDevice));

    checkCudaErrors(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
     

    // allocate chemical fields and density profile on CPU
    chemical->W_sp=dmatrix(0,chemical->N_spe-1,0,chemical->Nxyz-1);
    chemical->R_sp=dmatrix(0,chemical->N_spe-1,0,chemical->Nxyz-1);
    // propagators are allocated on GPU gobal memeory
    hipEventRecord(start, 0);
    checkCudaErrors(hipMallocManaged(&chain->qf, sizeof(double)* chain->Nxyz*chain->Ns));
    checkCudaErrors(hipMallocManaged(&chain->qb, sizeof(double)* chain->Nxyz*chain->Ns));

    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for the kernel: %f ms\n", time);

}

extern void field_cp_gpu(GRID *grid,CHEMICAL *chemical,CHAIN *chain) {
     int Nx,Ny,Nz,i,j,k,spe;
     long ijk;
     Nx=grid->Nx;   
     Ny=grid->Ny;   
     Nz=grid->Nz;   
     double ds;
     ds=1.0/chain->Ns;
      for (spe=0;spe<chemical->N_spe;spe++) {
	for(k=0;k<Nz;k++) {
	   for(j=0;j<Ny;j++){
	      for(i=0;i<Nx;i++){
		ijk=(long)((k*Ny+j)*Nx+i+ spe*chemical->Nxyz );// x is the fastest dimension!!
		chemical->exp_w[ijk]=exp(-0.5*ds*chemical->W_sp[spe][ijk]);
	                       }
                             }
                           }
	                 }

    checkCudaErrors(hipMemcpy(chemical->exp_w_cu, chemical->exp_w,sizeof(double)*chemical->Nxyz*chemical->N_spe,hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

}

extern void test_cufft(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){
       double *h_in;
       double complex *h_out;
       hipfftDoubleReal *d_in;
       hipfftDoubleComplex *d_out;
       int Nx=cufft_info->Nx;
       int Ny=cufft_info->Ny;
       int Nz=cufft_info->Nz;
       h_in = (double*) malloc(sizeof(double) * Nx*Ny*Nz);
       h_out = (double complex*) malloc(sizeof(double complex) * Nx*Ny*(Nz/2+1));   
       unsigned int in_mem_size = Nx*Ny*Nz*sizeof(hipfftDoubleReal);
       unsigned int out_mem_size = Nx*Ny*(Nz/2 + 1)*sizeof(hipfftDoubleComplex);
       checkCudaErrors(hipMalloc((void **)&d_in, in_mem_size));
       checkCudaErrors(hipMalloc((void **)&d_out, out_mem_size));
       int i,j,k,ijk;
       for (i=0, ijk=0; i < Nx; i++){
          for (j=0; j < Ny; j++){
             for (k=0; k < Nz; k++){
                 h_in[ijk]=(i+j+k)*1.0;
                 ijk++;
                                   }
                                 }
                              }

       checkCudaErrors(hipMemcpy(d_in, h_in, in_mem_size, hipMemcpyHostToDevice));
       if (hipfftExecD2Z(cufft_info->plan_forward[0], d_in, d_out) != HIPFFT_SUCCESS){ 
         fprintf(stderr, "CUFFT Error: Unable to execute plan\n"); 
                                                 return;   } 
       if (hipfftExecZ2D(cufft_info->plan_backward[0], d_out, d_in) != HIPFFT_SUCCESS){ 
         fprintf(stderr, "CUFFT Error: Unable to execute plan\n"); 
                                                 return;   } 
       if (hipDeviceSynchronize() != hipSuccess){ 
       fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
                                          return; }
       checkCudaErrors(hipMemcpy(h_out, d_out, out_mem_size, hipMemcpyDeviceToHost));
       checkCudaErrors(hipMemcpy(h_in, d_in, in_mem_size, hipMemcpyDeviceToHost));
       printf("h_out[0,0,0]:  = %.2f %+.2fi\n", creal(h_out[0]), cimag(h_out[0]));
       printf("h_in[0,0,1]:  = %.2f\n", h_in[1]/(Nx*Ny*Nz));


}


extern void finalize_cufft(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){
	int i;  //
	//int can_access_peer_0_1;
	
	//! free memery on GPU
	

	
	for (i=0; i < gpu_info->GPU_N; i++){
		
		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[i]));

		checkCudaErrors(hipfftDestroy(cufft_info->plan_forward[i]));
		checkCudaErrors(hipfftDestroy(cufft_info->plan_backward[i]));
		
		checkCudaErrors(hipFree(cufft_info->device_in[i]));
		checkCudaErrors(hipFree(cufft_info->device_out[i]));
	
		hipDeviceSynchronize();
	
	}
	
	//! free memery on CPU
	
	free(gpu_info->stream);
	free(gpu_info->GPU_list);

	printf("Wonderful We have successfully evaculate all the memery on GPU and CPU \n");
	hipDeviceReset();
}



