
#include"struct.h"

#include "init_cuda.h"

#include "cuda_aid.cuh"
#include <complex.h>

extern void init_cuda(GPU_INFO *gpu_info,int display){
	
	int gpu_count;
	//hipDeviceProp_t prop[64];
	int *gpuid;
	int i;
	gpu_count=0;
	
	gpuid=(int*)malloc(sizeof(int));

	
	//checkCudaErrors(hipGetDeviceCount(&gpu_info->GPU_N));

	//if(gpu_info->GPU_N==8) gpu_info->GPU_N=4;//! Set the number of GPU

		
	
	gpu_info->GPU_list=(int*)malloc(sizeof(int)*(gpu_info->GPU_N));

	for(i=0;i<(gpu_info->GPU_N);i++) {
		gpu_info->GPU_list[i]=i;}	//!Define on these GPU to calculate 

	
	for (i=0; i < gpu_info->GPU_N; i++){
		
        	checkCudaErrors(hipGetDeviceProperties(&gpu_info->prop[i], i)); // get the device properties for the specified device number i
		
		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[i])); // cuda runtime API, thread-safe, to select which GPU to execute CUDA calls on
		
		// Only boards based on Fermi can support P2P
		
            	gpuid[gpu_count++] = gpu_info->GPU_list[i];
                printf("Device Number: %d\n", i);
                printf("  Device name: %s\n", gpu_info->prop[i].name);
                printf("  Memory Clock Rate (KHz): %d\n",\
                       gpu_info->prop[i].memoryClockRate);
                printf("  Memory Bus Width (bits): %d\n",\
                       gpu_info->prop[i].memoryBusWidth);
                printf("  Peak Memory Bandwidth (GB/s): %f\n\n",\
                       2.0*gpu_info->prop[i].memoryClockRate*(gpu_info->prop[i].memoryBusWidth/8)/1.0e6);

                printf("Total global memory:           %lu\n",  gpu_info->prop[i].totalGlobalMem);
                printf("Total shared memory per block: %lu\n",  gpu_info->prop[i].sharedMemPerBlock);
                printf("Total registers per block:     %d\n",  gpu_info->prop[i].regsPerBlock);
                printf("Warp size:                     %d\n",  gpu_info->prop[i].warpSize);
                printf("Maximum memory pitch:          %lu\n",  gpu_info->prop[i].memPitch);
                printf("Maximum threads per block:     %d\n",  gpu_info->prop[i].maxThreadsPerBlock);
                for (int j = 0; j < 3; ++j)
                    printf("Maximum dimension %d of block:  %d\n", j, gpu_info->prop[i].maxThreadsDim[j]);
                for (int j = 0; j < 3; ++j)
                    printf("Maximum dimension %d of grid:   %d\n", j, gpu_info->prop[i].maxGridSize[j]);



		
		//gpu_info->thread=gpu_info->prop[i].maxThreadsPerBlock;
		
}

}


extern void initialize_cufft(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){

	
	int Dim[3];
	int i;
	int rank = 3;
	int Nx=cufft_info->Nx;
	int Ny=cufft_info->Ny;
	int Nz=cufft_info->Nz;
	
	long NxNyNz=Nx*Ny*Nz;  //,ijk;
	
	cufft_info->NxNyNz=NxNyNz;
	cufft_info->Nxh1=Nx/2+1; // R2C, first dimension is cut in half for reduandency
	cufft_info->Nxh1NyNz=cufft_info->Nxh1*Ny*Nz; // only near half size of grid in complex fft space
	int batch=1;
	
        // doing the factor decompose to determine the thread grid dimension?
	//printf("gpu_info->thread_sur %d\n",gpu_info->thread_sur);
	
	//char comment[200];

	
	//!----------- Initialize GPU memery settings. ------------------------------------------------------	
	
	//int nGPUs = gpu_info->GPU_N;
	
	cufft_info->device_in.resize(gpu_info->GPU_N);
	cufft_info->device_out.resize(gpu_info->GPU_N);
	//-----------! Initialize CUFFT settings. ------------------------------------------------------
	
	dim3 grid(cufft_info->Nx,cufft_info->Ny,cufft_info->Nz),block(1,1,1); // set the grid dimension as (Nx,Ny,Nz). one thread per block
	
	Dim[0]=Nz;Dim[1]=Ny;Dim[2]=Nx;

	cufft_info->plan_forward=(hipfftHandle *)malloc(sizeof(hipfftHandle)*gpu_info->GPU_N);
	cufft_info->plan_backward=(hipfftHandle *)malloc(sizeof(hipfftHandle)*gpu_info->GPU_N);

	for(int gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
		
		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[gpu_index]));
	
		checkCudaErrors(hipfftCreate(&cufft_info->plan_forward[gpu_index]));
		checkCudaErrors(hipfftCreate(&cufft_info->plan_backward[gpu_index]));
		
		
		if(rank==3){
			
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_forward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_D2Z, batch));
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_backward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_Z2D, batch));
		
		}
		else if(rank==2) {
		
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_forward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_D2Z, batch));
			checkCudaErrors(hipfftPlanMany (&cufft_info->plan_backward[gpu_index], rank, Dim, NULL, 1, 1, NULL, 1, 1, HIPFFT_Z2D, batch));

		}
	}
	
	
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed [  ]");
	printf("Wonderful We have successfully initialized cufft setting.\n");

	//-----------! Initialize malloc and initilize on CPU. ------------------------------------------------------	
	gpu_info->stream=(hipStream_t*)malloc( sizeof(hipStream_t)*gpu_info->GPU_N);
	
	
	printf("Wonderful We have successfully initialized CPU setting.\n");
	
	//-----------! Initialize malloc and initilize on each GPUs. ------------------------------------------------------	

	for (i=0; i < gpu_info->GPU_N; i++){

		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[i]));
		checkCudaErrors(hipStreamCreate(&gpu_info->stream[i]));
		checkCudaErrors(hipfftSetStream(cufft_info->plan_forward[i], gpu_info->stream[i]));
		checkCudaErrors(hipfftSetStream(cufft_info->plan_backward[i], gpu_info->stream[i]));
		
	//	checkCudaErrors(hipMallocManaged((void**)&(cufft_info->kxyzdz_cu[i]), sizeof(double)* NxNyNz));
		checkCudaErrors(hipMalloc(&(cufft_info->device_in[i]), sizeof(double)* cufft_info->NxNyNz*batch));
		checkCudaErrors(hipMalloc(&(cufft_info->device_out[i]), sizeof(hipfftDoubleComplex)* cufft_info->Nxh1NyNz*batch));

		checkCudaErrors(hipDeviceSynchronize());
		
		
	}
	
	
	
	printf("Wonderful We have successfully initialized all the data.\n");
	
	
}


extern void test_cufft(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){
       double *h_in;
       double complex *h_out;
       hipfftDoubleReal *d_in;
       hipfftDoubleComplex *d_out;
       int Nx=cufft_info->Nx;
       int Ny=cufft_info->Ny;
       int Nz=cufft_info->Nz;
       h_in = (double*) malloc(sizeof(double) * Nx*Ny*Nz);
       h_out = (double complex*) malloc(sizeof(double complex) * Nx*Ny*(Nz/2+1));   
       unsigned int in_mem_size = Nx*Ny*Nz*sizeof(hipfftDoubleReal);
       unsigned int out_mem_size = Nx*Ny*(Nz/2 + 1)*sizeof(hipfftDoubleComplex);
       checkCudaErrors(hipMalloc((void **)&d_in, in_mem_size));
       checkCudaErrors(hipMalloc((void **)&d_out, out_mem_size));
       int i,j,k,ijk;
       for (i=0, ijk=0; i < Nx; i++){
          for (j=0; j < Ny; j++){
             for (k=0; k < Nz; k++){
                 h_in[ijk]=(i+j+k)*1.0;
                 ijk++;
                                   }
                                 }
                              }

       checkCudaErrors(hipMemcpy(d_in, h_in, in_mem_size, hipMemcpyHostToDevice));
       if (hipfftExecD2Z(cufft_info->plan_forward[0], d_in, d_out) != HIPFFT_SUCCESS){ 
         fprintf(stderr, "CUFFT Error: Unable to execute plan\n"); 
                                                 return;   } 
       if (hipfftExecZ2D(cufft_info->plan_backward[0], d_out, d_in) != HIPFFT_SUCCESS){ 
         fprintf(stderr, "CUFFT Error: Unable to execute plan\n"); 
                                                 return;   } 
       if (hipDeviceSynchronize() != hipSuccess){ 
       fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
                                          return; }
       checkCudaErrors(hipMemcpy(h_out, d_out, out_mem_size, hipMemcpyDeviceToHost));
       checkCudaErrors(hipMemcpy(h_in, d_in, in_mem_size, hipMemcpyDeviceToHost));
       printf("h_out[0,0,0]:  = %.2f %+.2fi\n", creal(h_out[0]), cimag(h_out[0]));
       printf("h_in[0,0,1]:  = %.2f\n", h_in[1]/(Nx*Ny*Nz));


}


extern void finalize_cufft(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){
	int i;  //
	//int can_access_peer_0_1;
	
	//! free memery on GPU
	

	
	for (i=0; i < gpu_info->GPU_N; i++){
		
		checkCudaErrors(hipSetDevice(gpu_info->GPU_list[i]));

		checkCudaErrors(hipfftDestroy(cufft_info->plan_forward[i]));
		checkCudaErrors(hipfftDestroy(cufft_info->plan_backward[i]));
		
		checkCudaErrors(hipFree(cufft_info->device_in[i]));
		checkCudaErrors(hipFree(cufft_info->device_out[i]));
	
		hipDeviceSynchronize();
	
	}
	
	
	
	//! free memery on CPU
	
	free(gpu_info->stream);
	free(gpu_info->GPU_list);

	printf("Wonderful We have successfully evaculate all the memery on GPU and CPU \n");
	hipDeviceReset();
}



